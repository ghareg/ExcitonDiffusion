#include "operator.h"
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>
#include <cmath>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>

int main(void)
{
	int Size = nkx * nky;
	sType inout(4 * Size);
	shType outCur(2 * Size);
	shType outTot(2 * Size);
	polaritonSchrodinger pSch(nkx, nky, 2, 2);
	observer obs(nkx, nky, outCur, tmax, tdif);

	thrust::fill(outTot.begin(), outTot.end(), 0.0f);
	thrust::plus<dtype> op;
	float tstart = 0.0f;

	boost::numeric::odeint::adams_bashforth_moulton<5 , sType, dtype, sType, dtype> abm;
	//boost::numeric::odeint::runge_kutta_dopri5< sType , dtype, sType , dtype > rkd;

	for (int count = 0; count < impl; ++count) {
		thrust::fill(inout.begin(), inout.end(), 0.0f);
		pSch.initUScat();
		abm.initialize(boost::ref(pSch), inout, tstart, tstep);
		size_t steps = boost::numeric::odeint::integrate_const(abm, boost::ref(pSch), inout, tstart, tmax - tdif, tstep);
	//size_t steps = boost::numeric::odeint::integrate_const(make_controlled( 1.0e-4, 1.0e-4, rkd), boost::ref(pSch), inout, tstart, tmax, tstep, boost::ref(obs));
		obs.initialize();
		float t = tmax - tdif;
		while(t < tmax) {
			steps = boost::numeric::odeint::integrate_const(abm, boost::ref(pSch), inout, t, t + tostep, tstep);
			obs(inout, t);
			t += tostep;
		}
		obs.normalize();
		//thrust::copy(inout.begin(), inout.end(), outCur.begin());
		thrust::transform(outTot.begin(), outTot.end(), outCur.begin(), outTot.begin(), op);
	}

	thrust::constant_iterator<dtype> iter(1.0f / impl);
	thrust::transform(outTot.begin(), outTot.end(), iter, outTot.begin(), thrust::multiplies<dtype>());

	for (int ix = 0; ix < nkx; ++ix) {
		for (int iy = 0; iy < nky; ++iy) {
			int idx = ix * nky + iy;
			int hkx = ix <= nkx / 2 ? 0 : nkx;
			int hky = iy <= nky / 2 ? 0 : nky;
			float x = (ix - hkx) * xstep;
			float y = (iy - hky) * ystep;
			//float p1r = sqrt2 * (outTot[2 * idx] + outTot[Size + 2 * idx]);
			//float p1i = sqrt2 * (outTot[2 * idx + 1] + outTot[Size + 2 * idx + 1]);
			//float p2r = sqrt2 * (-outTot[2 * idx + 1] + outTot[Size + 2 * idx + 1]);
			//float p2i = sqrt2 * (outTot[2 * idx] - outTot[Size + 2 * idx]);
			//float phi1Norm = outTot[2 * idx] * outTot[2 * idx] + outTot[2 * idx + 1] * outTot[2 * idx + 1]; 
			//float phi2Norm = outTot[Size + 2 * idx] * outTot[Size + 2 * idx] + outTot[Size + 2 * idx + 1] * outTot[Size + 2 * idx + 1];
			std::printf("%.6f\t%.6f\t%.6f\t%.6f\n", x, y, outTot[idx], outTot[Size + idx]); //p1r * p1r + p1i * p1i, p2r * p2r + p2i * p2i); //phi1Norm, phi2Norm);
		}
	}
}

