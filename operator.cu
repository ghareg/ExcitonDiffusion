#include "hip/hip_runtime.h"
#include "operator.h"
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <thrust/iterator/constant_iterator.h>

polaritonSchrodinger::polaritonSchrodinger(Count xSize, Count ySize, int rank, int batch): xSize_(xSize), ySize_(ySize), Size_(xSize_ * ySize_), Sizem1_(1.0f / Size_), rank_(rank), batch_(batch), xT_(2 * batch_ * Size_), n_(new Count[rank_]), tex_(), Sex_(), pump_(), UScat_(Size_), cr_(Size_), UScatl_(Size_), rng_(), dist_(0.0f, 1.0f){
	n_[0] = xSize;
	n_[1] = ySize;
	createMat();
	hipfftPlanMany(&plan_, rank_, n_, NULL, 1, Size_, NULL, 1, Size_, HIPFFT_C2C, batch_);
}

void polaritonSchrodinger::operator() (const sType& x, sType& dxdt, dtype t)
{
	hipfftComplex* xc = reinterpret_cast<hipfftComplex*>(const_cast<dtype*>(thrust::raw_pointer_cast(x.data())));
	hipfftComplex* xTc_ = reinterpret_cast<hipfftComplex*>(const_cast<dtype*>(thrust::raw_pointer_cast(xT_.data())));
	hipfftExecC2C(plan_, xc,  xTc_, HIPFFT_FORWARD);

	pType phi1It(reinterpret_cast<Compl*>(thrust::raw_pointer_cast(xT_.data())));
	pType phi2It = phi1It + Size_;
	pType xTend  = phi2It + Size_;

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(phi1It, phi2It, tex_.begin(),
					Sex_.begin(), phi1It, phi2It)), 
			thrust::make_zip_iterator(thrust::make_tuple(phi2It, xTend, tex_.end(),
					Sex_.end(), phi2It, xTend)), KSysFunctor());

	hipfftComplex* dxdtc = reinterpret_cast<hipfftComplex*>(const_cast<dtype*>(thrust::raw_pointer_cast(dxdt.data())));

	hipfftExecC2C(plan_, xTc_, dxdtc, HIPFFT_BACKWARD);

	cpType cphi1It(reinterpret_cast<const Compl*>(thrust::raw_pointer_cast(x.data())));
	cpType cphi2It = cphi1It + Size_;
	cpType xend  = cphi2It + Size_;

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(cphi1It, cphi2It, pump_.begin(), UScat_.begin(),
					phi1It, phi2It)),
			thrust::make_zip_iterator(thrust::make_tuple(cphi2It, xend, pump_.end(), UScat_.end(),
					phi2It, xTend)), XYSysFunctor(CXLft));
	
	thrust::transform(xT_.begin(), xT_.end(), dxdt.begin(), dxdt.begin(), SumMultFunctor(Sizem1_));
}


polaritonSchrodinger::KSysFunctor::KSysFunctor()
{
}

template<class Tuple>
 __host__ __device__
void polaritonSchrodinger::KSysFunctor::operator()(Tuple t)
{
	Compl p1;
	Compl p2;
	p1.real = thrust::get<2>(t) * thrust::get<0>(t).imag + thrust::get<3>(t).real * thrust::get<1>(t).imag + thrust::get<3>(t).imag * thrust::get<1>(t).real;
	p1.imag = -thrust::get<2>(t) * thrust::get<0>(t).real - thrust::get<3>(t).real * thrust::get<1>(t).real + thrust::get<3>(t).imag * thrust::get<1>(t).imag;
	p2.real = thrust::get<2>(t) * thrust::get<1>(t).imag + thrust::get<3>(t).real * thrust::get<0>(t).imag - thrust::get<3>(t).imag * thrust::get<0>(t).real;
	p2.imag = -thrust::get<2>(t) * thrust::get<1>(t).real - thrust::get<3>(t).real * thrust::get<0>(t).real - thrust::get<3>(t).imag * thrust::get<0>(t).imag;
	thrust::get<4>(t) = p1;
	thrust::get<5>(t) = p2;
}

polaritonSchrodinger::XYSysFunctor::XYSysFunctor(float CXLft): 
	CXLft_(CXLft) 
{
}

template<class Tuple>
 __host__ __device__
void polaritonSchrodinger::XYSysFunctor::operator()(Tuple t)
{
	Compl p1;
	Compl p2;

	p1.real = CXLft_ * thrust::get<0>(t).real + thrust::get<2>(t).imag/* + thrust::get<2>(t).real*/ + thrust::get<3>(t) * thrust::get<0>(t).imag;
	p1.imag = CXLft_ * thrust::get<0>(t).imag - thrust::get<2>(t).real/* + thrust::get<2>(t).imag*/ - thrust::get<3>(t) * thrust::get<0>(t).real;
	p2.real = CXLft_ * thrust::get<1>(t).real /*+ thrust::get<2>(t).imag - thrust::get<2>(t).real*/ + thrust::get<3>(t) * thrust::get<1>(t).imag;
	p2.imag = CXLft_ * thrust::get<1>(t).imag /*- thrust::get<2>(t).real - thrust::get<2>(t).imag*/ - thrust::get<3>(t) * thrust::get<1>(t).real;
	thrust::get<4>(t) = p1;
	thrust::get<5>(t) = p2;
}

void polaritonSchrodinger::createMat()
{
	std::vector<dtype> tex(Size_);
	std::vector<Compl> Sex(Size_);
	thrust::host_vector<Compl> pump(Size_);
	const int hxSize = xSize_ / 2;
	const int hySize = ySize_ / 2;
	#pragma omp parallel for
	for(int ikx = 0; ikx < xSize_; ++ikx) {
		for (int iky = 0; iky < ySize_; ++iky) {
			int hnkx = ikx <= hxSize ? 0 : xSize_;
			int hnky = iky <= hySize ? 0 : ySize_;
			float kx = (ikx - hnkx) * kxstep;
			float ky = (iky - hnky) * kystep;
			float ks = kx * kx + ky * ky;
			float k = std::sqrt(ks);
			int idx = ikx * ySize_ + iky;
			float CTex = Ex0 + h2mMx * ks; // + alpha * k;
			float km1 = 1.0f / k;
			if (ks > 1E-8) {
				Sex[idx].real = /*alpha * (kx * kx - ky * ky) * km1 * hbarm1;*/ beta * (kx * kx - ky * ky) * hbarm1;
				Sex[idx].imag = /*-2.0f * alpha * kx * ky * km1 * hbarm1;*/ -2.0 * beta * kx * ky * hbarm1;
			}
			else {
				Sex[idx].real = 0.0f;
				Sex[idx].imag = 0.0f;
			}
			tex[idx] = CTex * hbarm1;
			float Pol = Ex0 + h2mMx * ks;
			float Norm = sqrt2 * P0 * expf(-(k - kpa) * (k - kpa) * sigma * sigma * 0.25f);
			Norm *= Gamma * hbarm1 / (Pol * Pol + Gamma * Gamma);
			pump[idx].real = -Norm * Gamma;
			pump[idx].imag = Norm * Pol;
		}
	}
	tex_ = tex;
	Sex_ = Sex;
	pump_ = pump;
	
	hipfftHandle plan;
	hipfftPlan2d(&plan, xSize_, ySize_, HIPFFT_C2C);
	hipfftComplex* pp = reinterpret_cast<hipfftComplex*>(const_cast<Compl*>(thrust::raw_pointer_cast(pump_.data())));
	hipfftExecC2C(plan, pp, pp, HIPFFT_BACKWARD);
	thrust::transform(pump_.begin(), pump_.end(), pump_.begin(), MultFunctor(Sizem1_));

/*	pump = pump_;
	for (int ix = 0; ix < nkx; ++ix) {
		for (int iy = 0; iy < nky; ++iy) {
			int idx = ix * nky + iy;
			int hkx = ix <= nkx / 2 ? 0 : xSize_;
			int hky = iy <= nky / 2 ? 0 : ySize_;
			float x = (ix - hkx) * xstep;
			float y = (iy - hky) * ystep;
			std::printf("%6f\t%6f\t%6f\n", x, y, pump[idx].real * pump[idx].real + pump[idx].imag * pump[idx].imag);
		}
	}*/
	hipfftDestroy(plan);
}

void polaritonSchrodinger::initUScat()
{
	for (int i = 0; i < Size_; ++i) {
		cr_[i] = dist_(rng_);
	}
	const Count hxSize = xSize_ / 2;
	const Count hySize = ySize_ / 2;
	#pragma omp parallel for 
	for(int ix = 0; ix < xSize_; ++ix) {
		for (int iy = 0; iy < ySize_; ++iy) {
			float result = 0.0f;
			int idx = ix * ySize_ + iy;
			int hkx = (ix <= hxSize) ? 0 : xSize_;
			int hky = (iy <= hySize) ? 0 : ySize_;
			float xc = (ix - hkx) * xstep;
			float yc = (iy - hky) * ystep;
			for(int ixp = 0; ixp < xSize_; ++ixp) {
				for (int iyp = 0; iyp < ySize_; ++iyp) {
					int idxp = ixp * ySize_ + iyp;
					float xcp = (ixp - hxSize) * xstep;
					float ycp = (iyp - hySize) * ystep;
					float dif = (xc - xcp) * (xc - xcp) + (yc - ycp) * (yc - ycp);
					result += cr_[idxp] * expf(-2.0f * dif / (lx * lx));
				}
			}
			UScatl_[idx] = 0.0; //2.0 * U0 * std::sqrt(xstep * ystep) * result / (hbar * lx * std::sqrt(Pi));
		}
	}
	/*for (int ix = 0; ix < nkx; ++ix) {
		for (int iy = 0; iy < nky; ++iy) {
			int idx = ix * nky + iy;
			int hkx = ix <= hxSize ? 0 : xSize_;
			int hky = iy <= hySize ? 0 : ySize_;
			float x = (ix - hkx) * xstep;
			float y = (iy - hky) * ystep;
			std::printf("%3f\t%3f\t%3f\n", x, y, thrust::abs(UScat[idx]));
		}
	}*/

	thrust::copy(UScatl_.begin(), UScatl_.end(), UScat_.begin());
	//UScat_ = UScatl_;
}

polaritonSchrodinger::MultFunctor::MultFunctor(dtype mult) : mult_(mult)
{
}

template<class Type>
__host__ __device__
Type polaritonSchrodinger::MultFunctor::operator()(const Type& x)
{
	Compl y = {mult_ * x.real, mult_ * x.imag};
	return y;
}

polaritonSchrodinger::SumMultFunctor::SumMultFunctor(dtype mult) : mult_(mult)
{
}

template<class Type>
__host__ __device__
Type polaritonSchrodinger::SumMultFunctor::operator()(const Type& x, const Type& y)
{
	return x + mult_ * y;
}

polaritonSchrodinger::~polaritonSchrodinger()
{
	delete[] n_;
	hipfftDestroy(plan_);
}

observer::observer(Count xSize, Count ySize, shType& xInt, dtype tmax, dtype tdif) :
   	xSize_(xSize), ySize_(ySize), xInt_(xInt), xTemp_(2 * xSize_ * ySize_), /*kTemp_(4 * xSize_ * ySize_),*/ xDTemp_(2 * xSize_ * ySize_), tmax_ (tmax), tdif_(tdif), steps_(0), op_()
{
	/*Count n[2];
	n[0] = xSize_;
	n[1] = ySize_;
	Count Size = xSize_ * ySize_;
	int rank = 2;
	int batch = 2;
	hipfftPlanMany(&plan_, rank, n, NULL, 1, Size, NULL, 1, Size, HIPFFT_C2C, batch);*/
}

observer::~observer()
{
	//hipfftDestroy(plan_);
}

void observer::initialize()
{
	thrust::fill(xInt_.begin(), xInt_.end(), 0.0f);
	steps_ = 0;
}


void observer::operator() (const sType& x, dtype t)
{
	const Count Size = nkx * nky;
//	thrust::copy(x.begin(), x.end(), xTemp_.begin());
/*	hipfftComplex* xd = reinterpret_cast<hipfftComplex*>(const_cast<dtype*>(thrust::raw_pointer_cast(x.data())));
	hipfftComplex* kd = reinterpret_cast<hipfftComplex*>(const_cast<dtype*>(thrust::raw_pointer_cast(kTemp_.data())));
	hipfftExecC2C(plan_, xd, kd, HIPFFT_FORWARD);*/

	cpType cphi1It(reinterpret_cast<const Compl*>(thrust::raw_pointer_cast(x.data())));
	cpType cphi2It = cphi1It + Size;
	cpType xend  = cphi2It + Size;

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(cphi1It, cphi2It, xDTemp_.begin(), xDTemp_.begin() + Size)), 
			thrust::make_zip_iterator(thrust::make_tuple(cphi2It, xend, xDTemp_.begin() + Size, xDTemp_.end())), OverFunctor(sqrt2));
/*	pType phi1It(reinterpret_cast<Compl*>(thrust::raw_pointer_cast(kTemp_.data())));
	pType phi2It = phi1It + Size;
	pType kend  = phi2It + Size;

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(phi1It, phi2It, xDTemp_.begin(), xDTemp_.begin() + Size)), 
			thrust::make_zip_iterator(thrust::make_tuple(phi2It, kend, xDTemp_.begin() + Size, xDTemp_.end())), OverFunctor(sqrt2));*/
	thrust::copy(xDTemp_.begin(), xDTemp_.end(), xTemp_.begin());
	thrust::transform(xInt_.begin(), xInt_.end(), xTemp_.begin(), xInt_.begin(), op_);
	steps_++;
}

void observer::normalize()
{
	thrust::constant_iterator<dtype> iter(1.0f / steps_);
	thrust::transform(xInt_.begin(), xInt_.end(), iter, xInt_.begin(), thrust::multiplies<dtype>());
}

observer::OverFunctor::OverFunctor(dtype sqrt2) : sqrt2_(sqrt2)
{
}
	
template<class Tuple>
 __host__ __device__
void observer::OverFunctor::operator()(Tuple t)
{
	Compl p1;
	Compl p2;

	p1.real = sqrt2_ * (thrust::get<0>(t).real + thrust::get<1>(t).real);
	p1.imag = sqrt2_ * (thrust::get<0>(t).imag + thrust::get<1>(t).imag);
	p2.real = sqrt2_ * (-thrust::get<0>(t).imag + thrust::get<1>(t).imag);
	p2.imag = sqrt2_ * (thrust::get<0>(t).real - thrust::get<1>(t).real);
	thrust::get<2>(t) = p1.real * p1.real + p1.imag * p1.imag;
	thrust::get<3>(t) = p2.real * p2.real + p2.imag * p2.imag;
}


